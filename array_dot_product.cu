#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void MatrixMulkernel(int *c, const int* a,const int* b,int width){
    int sum=0;
    int x,y,i;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    x = blockIdx.x * blockDim.x + threadIdx.x;

    for(int k=0; k<width; k++){
       sum += a[y* width + k] * b[k* width +x];
    }
    i = y * width + x;
    c[i] = sum;
   
}

void PrintMatrix(int a[][16], int size){
    for(int i=0; i<size; i++){
        printf("|");
        for(int j=0; j<size; j++){
            printf("%6d",a[i][j]);
            if(j==7) printf("|      |");
        }
        
        printf("|\n");
        if(i==7) printf("\n");
    }
}

int main(){
    const int WIDTH = 16;
    const int TILE_WIDTH = 2;
    int a[WIDTH][WIDTH];
    int b[WIDTH][WIDTH];
    int c[WIDTH][WIDTH] = {0,};

    for(int y=0; y<WIDTH; y++){
        for(int x=0; x<WIDTH; x++){
            a[y][x] = y+x;
            b[y][x] = y*10+x; 
        }
    }

    int *dev_a, *dev_b, *dev_c=0;
    hipMalloc((void**)&dev_a, WIDTH*WIDTH*sizeof(int));
    hipMalloc((void**)&dev_b, WIDTH*WIDTH*sizeof(int));
    hipMalloc((void**)&dev_c, WIDTH*WIDTH*sizeof(int));

    hipMemcpy(dev_a, a, WIDTH*WIDTH*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, WIDTH*WIDTH*sizeof(int), hipMemcpyHostToDevice);
    dim3 dimGrid(WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH,1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH,1);

    MatrixMulkernel<<<dimGrid,dimBlock>>>(dev_c,dev_a,dev_b,WIDTH);
    hipDeviceSynchronize();

    hipMemcpy(c,dev_c,WIDTH*WIDTH*sizeof(int),hipMemcpyDeviceToHost);
    printf("<Matrix a> \n");
    PrintMatrix(a,WIDTH);printf("\n\n");
    printf("<Matrix b> \n");
    PrintMatrix(b,WIDTH);printf("\n\n");
    printf("<Matrix c> \n");
    PrintMatrix(c,WIDTH);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
